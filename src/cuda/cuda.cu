#include <stdio.h>
#include <iostream>
#include <ctime>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iomanip>

__constant__ int sqrtThreadsPerBlock;

__global__ void makeRightHandSideIdentity(double *mat, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        for (int j = n; j < 2 * n; ++j) {
            if (tid == j - n) {
                mat[tid * (2 * n) + j] = 1.0;
            } else {
                mat[tid * (2 * n) + j] = 0.0;
            }
        }
    }
}

__global__ void partialPivoting(double *mat, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n - 1) {
        if (mat[tid * (2 * n) + 1] > mat[(tid + 1) * (2 * n) + 1]) {
            for (int j = 0; j < 2 * n; ++j) {
                double temp = mat[tid * (2 * n) + j];
                mat[tid * (2 * n) + j] = mat[(tid + 1) * (2 * n) + j];
                mat[(tid + 1) * (2 * n) + j] = temp;
            }
        }
    }
}

__global__ void reduceToDiagonal(double *mat, int n, int currentRow) {
    __shared__ double d;
    int tid = (blockIdx.y * sqrtThreadsPerBlock * sqrtThreadsPerBlock) + (threadIdx.x * sqrtThreadsPerBlock + threadIdx.y);
    if (blockIdx.x != currentRow) {
        if (threadIdx.x + threadIdx.y == 0)
        {
            d = mat[blockIdx.x * 2*n + currentRow] / mat[currentRow * 2*n + currentRow];
        }
        __syncthreads();
        mat[blockIdx.x * 2*n + tid] -= mat[currentRow * 2*n + tid] * d;
    }
}

__global__ void reduceToUnitMatrix(double *mat, int n) {
    __shared__ double d;
    int tid = (blockIdx.y * sqrtThreadsPerBlock * sqrtThreadsPerBlock) + (threadIdx.x * sqrtThreadsPerBlock + threadIdx.y);
    if (mat[blockIdx.x * 2*n + blockIdx.x] != 0) {
        if (threadIdx.x + threadIdx.y == 0)
        {
            d = mat[blockIdx.x * 2*n + blockIdx.x];
        }
        __syncthreads();
        mat[blockIdx.x * 2*n + tid] /= d;
    }
}

void printResultMatrix(double *mat, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = n; j < 2 * n; ++j) {
            std::cout << std::setw(8) << mat[i * (2 * n) + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    int n;
    double *mat = nullptr;

    std::cin >> n;

    // Allocate ukuran matriks
    mat = new double[2 * n * 2 * n];

    // Input nilai dalam matriks
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cin >> mat[i * (2 * n) + j];
        }
    }

    // CUDA memory allocation ke GPU
    double *d_mat;
    hipMalloc((void **)&d_mat, (2 * n) * (2 * n) * sizeof(double));
    hipMemcpy(d_mat, mat, (2 * n) * (2 * n) * sizeof(double), hipMemcpyHostToDevice);

    // Launch CUDA kernels
    int tpb = static_cast<int>(sqrt(n/2));
    hipMemcpyToSymbol(HIP_SYMBOL(sqrtThreadsPerBlock), &tpb, sizeof(tpb));
    dim3 threadsPerBlock(tpb, tpb);
    dim3 numBlocks(n, 4);

    // Right hand side identity
    makeRightHandSideIdentity<<<numBlocks, threadsPerBlock>>>(d_mat, n);
    hipDeviceSynchronize();

    // Partial Pivoting
    for (int i = n; i > 1; --i) {
        if (mat[2*n*(i - 1)+1] < mat[2*n*i+1]) {
            for (int j = 0; j < 2 * n; ++j) {
                double d = mat[2*n*i+j];
                mat[2*n*i+j] = mat[2*n*(i - 1)+j];
                mat[2*n*(i - 1)+j] = d;
            }
        }
    }

    // Reduce to Diagonal Matrix
    for (int i = 0; i < n; ++i){
        if (mat[i*2*n + i] != 0){
            reduceToDiagonal<<<numBlocks, threadsPerBlock>>>(d_mat, n, i);
        }
        hipDeviceSynchronize();
    }

    // Reduce to Unit Matrix
    reduceToUnitMatrix<<<numBlocks, threadsPerBlock>>>(d_mat, n);
    hipDeviceSynchronize();

    // Copy hasil balik ke CPU
    hipMemcpy(mat, d_mat, (2 * n) * (2 * n) * sizeof(double), hipMemcpyDeviceToHost);

    // Print output matrix
    std::cout << "Output matrix:" << std::endl;
    printResultMatrix(mat, n);

    // Free memory
    delete[] mat;
    hipFree(d_mat);

    return 0;
}